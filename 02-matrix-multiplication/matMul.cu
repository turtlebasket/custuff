#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matmul(float** A, float** B, float** C) {
    int bix = blockIdx.x;
    int tix = threadIdx.x;
}

void print_matrix(float** m, size_t rows, size_t cols, char* name) {
    std::cout << "==== MATRIX: " << name << " ====\n";
    for (int r = 0; r < rows; r++) {
        for (int c = 0; c < cols; c++) {
            std::cout << m[r][c] << "\t";
        }
        std::cout << "\n"
    }
}

int main() {

    return 0;
}
