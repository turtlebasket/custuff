#include "hip/hip_runtime.h"
#include "matrixHelpers.hpp"
#include <hip/hip_runtime.h>

#define ROWS_A 9
#define COLS_A 7
#define N_THREADS (ROWS_A * ROWS_A)

__global__ void matmul(float** A, float** B, float** C) {
    int bix = blockIdx.x;
    int tix = threadIdx.x;

    // __shared__ float tile_A[]
    // __syncthreads();
}

int main() {
    float** A = gen_row_inc_matrix(ROWS_A, COLS_A);
    float** B = gen_row_inc_matrix(COLS_A, ROWS_A);
    float** C;
    print_matrix(A, ROWS_A, COLS_A, "A");
    print_matrix(B, COLS_A, ROWS_A, "B");

    // hipMemcpyAsync()

    return 0;
}
