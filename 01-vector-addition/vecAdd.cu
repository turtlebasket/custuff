/**
* Basic example; divides a n-element vector add operation two different ways:
* - 1 block, N threads per block
* - N blocks, 1 thread per block
*/

#include <iostream>
#include <hip/hip_runtime.h>

#define VEC_SIZE 8

/**
 * @brief Vector addition using 1 block and N threads per block
 * 
 * @param A Input Vector pointer 1
 * @param B Input Vector pointer 2
 * @param C Result Vector pointer
 */
__global__ void vecAdd_A(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

/**
 * @brief Vector addition using N block and 1 thread per block
 * 
 * @param A Input Vector pointer 1
 * @param B Input Vector pointer 2
 * @param C Result Vector pointer
 */
__global__ void vecAdd_B(float* A, float* B, float* C)
{
    int i = blockIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    float a[VEC_SIZE] = {1., 2., 3., 4., 5., 6., 7., 8.};
    float b[VEC_SIZE] = {8., 7., 28., 1., 6., 2., 3., 5.};
    size_t vec_bytes = VEC_SIZE * sizeof(float);
    float *c = (float *)malloc(vec_bytes);
    float *d = (float *)malloc(vec_bytes);
    float *d_a, *d_b, *d_c, *d_d;

    // allocate memory on device
    hipMalloc((void **) &d_a, vec_bytes);
    hipMalloc((void **) &d_b, vec_bytes);
    hipMalloc((void **) &d_c, vec_bytes);
    hipMalloc((void **) &d_d, vec_bytes);

    // copy input vectors from host memory to device
    hipMemcpy(d_a, &a, vec_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, vec_bytes, hipMemcpyHostToDevice);

    // run vecAdd kernel with 8 threads
    vecAdd_A<<<1, VEC_SIZE>>>(d_a, d_b, d_c);
    vecAdd_B<<<VEC_SIZE, 1>>>(d_a, d_b, d_d);

    // copy result vectors from device memory back to host
    hipMemcpy(c, d_c, vec_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(d, d_d, vec_bytes, hipMemcpyDeviceToHost);

    // print result C
    std::cout << "C: ";
    for (int i = 0; i < VEC_SIZE; i++) {
        std::cout << *(c+i) << " ";
    }
    std::cout << std::endl;

    // print result D
    std::cout << "D: ";
    for (int i = 0; i < VEC_SIZE; i++) {
        std::cout << *(d+i) << " ";
    }
    std::cout << std::endl;

    // free device memory
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);
    hipFree(d_d);

    return 0;
}
