#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matrixHelpers.hpp"

#define ROWS_A 9
#define COLS_A 7
#define N_THREADS (ROWS_A * ROWS_A)

__global__ void genIncMatrix(float* mat) {

}

__global__ void matmul(float* A, float* B, float* C) {
    int bix = blockIdx.x;
    int tix = threadIdx.x;

    // __shared__ float tile_A[]
    // __syncthreads();
}

int main() {
    float** A = gen_row_inc_matrix(ROWS_A, COLS_A);
    float** B = gen_row_inc_matrix(COLS_A, ROWS_A);
    float** C;
    float** d_A, d_B, d_C;
    size_t matrix_bytes = ROWS_A * sizeof(float*);

    print_matrix(A, ROWS_A, COLS_A, "A");
    print_matrix(B, COLS_A, ROWS_A, "B");

    hipMalloc((void **) d_A, matrix_bytes);
    hipMalloc((void **) d_B, matrix_bytes);

    hipMemcpyAsync(d_A, A, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_B, B, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
