/**
* Basic example; divides a n-element vector add operation two different ways:
* - 1 block, N threads per block
* - N blocks, 1 thread per block
*/

#include <iostream>
#include <hip/hip_runtime.h>

#define VEC_SIZE 102400
#define THREADS_PER_BLOCK 1024
#define VEC_BYTES (VEC_SIZE * sizeof(float))

__global__ void genIncVector(float *V) {
    int i = (blockIdx.x * THREADS_PER_BLOCK) + threadIdx.x;
    V[i] = i + 1;
}

__global__ void vecAdd(float* A, float* B, float* C) {
    int i = (blockIdx.x * THREADS_PER_BLOCK) + threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    float *a = (float *)malloc(VEC_BYTES);
    float *d_a, *d_b;

    // allocate memory on device
    hipMalloc((void **) &d_a, VEC_BYTES);
    hipMalloc((void **) &d_b, VEC_BYTES);

    genIncVector<<<VEC_SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a);
    genIncVector<<<VEC_SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_b);

    // run vecAdd kernel with 8 threads, N_ITERATIONS times
    // Basic operation: A += B
    vecAdd<<<VEC_SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_a);

    // copy result vectors from device memory back to host
    hipMemcpy(a, d_a, VEC_BYTES, hipMemcpyDeviceToHost);

    // print result C
    // std::cout << "A: ";
    // for (int i = 0; i < VEC_SIZE; i++) {
    //     std::cout << *(a+i) << " ";
    // }
    // std::cout << std::endl;

    // free device memory
    hipFree(d_a); 
    hipFree(d_b); 

    return 0;
}
